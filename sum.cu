#include <hip/hip_runtime.h>
#include <iostream>


__global__ void sum(double *d_a, double *d_b, int n)
{
    int idx = threadIdx.x;
    int i = 2, j = 1;
    do {
        if (idx % i == 0)
        {
            d_a[idx] += d_a[idx + j];
        }
        i *= 2;
        j *= 2;
    } while (n /= 2);
    d_b[0] = d_a[0];
}


int main(void)
{
    int n{5000};
    double *h_a = (double *) malloc(sizeof(double) * n);
    double *h_b = (double *) malloc(sizeof(double));
    double *d_a;
    double *d_b;
    hipMalloc(&d_a, sizeof(double) * n);
    hipMalloc(&d_b, sizeof(double));

    for(int i=0; i < n; i++)
    {
        h_a[i] = i + 1;
    }
    h_b[0] = 0;
    hipMemcpy(d_a, h_a, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(double), hipMemcpyHostToDevice);
    sum<<<1, n>>>(d_a, d_b, n);
    hipMemcpy(h_b, d_b, sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "The result is " << h_b[0] << std::endl;
    free(h_a);
    free(h_b);
    hipFree(d_a);
    hipFree(d_b);

}
