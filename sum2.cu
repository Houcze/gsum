#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

__global__ void l_trans(double *arr, double *result, int length, int i)
{
    int idx = threadIdx.x;
    if (idx < length - i)
    {
        result[idx] = arr[idx + i];
    }
}

__global__ void sum(double *a, double *b, int length)
{
    int idx = threadIdx.x;
    if (idx < length)
    {
        a[idx] = a[idx] + b[idx];
    }
}

double sum(double *h_a, int n)
{
    double *h_b;
    double *d_a;
    double *d_b;
    h_b = (double *)malloc(sizeof(double) * n);
    memset(h_b, sizeof(double) * n, 0);

    hipMalloc(&d_a, sizeof(double) * n);
    hipMalloc(&d_b, sizeof(double) * n);
    hipMemcpy(d_a, h_a, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(double) * n, hipMemcpyHostToDevice);

    double *d_tmp;
    hipMalloc(&d_tmp, sizeof(double) * n);

    for (int i = 0; i < n; i++)
    {
        hipMemset(d_tmp, sizeof(double) * n, 0);
        l_trans<<<ceil(n / 128), 128>>>(d_a, d_tmp, n, i);
        sum<<<ceil(n / 128), 128>>>(d_b, d_tmp, n);
    }
    hipMemcpy(h_b, d_b, sizeof(double), hipMemcpyDeviceToHost);
    // std::cout << h_b[0] << std::endl;
    double result = h_b[0];
    free(h_b);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_tmp);
    return result;    
}

int main(void)
{
    double *h_a;
    int n = 50000;
    h_a = (double *)malloc(sizeof(double) * n);
    for (int i = 0; i < n; i++)
    {
        h_a[i] = i + 1;
    }

    std::cout << sum(h_a, n) << std::endl;

}